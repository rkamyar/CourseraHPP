#include "hip/hip_runtime.h"
#include    <wb.h>

#define HISTOGRAM_LENGTH 256


__global__ void convertToChar(float * input, unsigned char * ucharInput, int width, int height)
{
	int bx = blockIdx.x;  int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	
	int row = by*blockDim.y+ty;
	int col = bx*blockDim.x+tx;
	int index = row*width + col;
	
	if(row < height && col < width)
	{
		ucharInput[index*3]   = (unsigned char) (255 * input[index*3]); //r
		ucharInput[index*3+1] = (unsigned char) (255 * input[index*3+1]); //g
		ucharInput[index*3+2] = (unsigned char) (255 * input[index*3+2]); //b
	}
	
	
}


__global__ void convertToGrayScale(unsigned char * ucharImg, unsigned char * grayImg, int width, int height)
{
	
	int bx = blockIdx.x;  int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	
	int row = by*blockDim.y+ty;
	int col = bx*blockDim.x+tx;
	int index = row*width + col;
	
	if(row < height && col < width)
	{
		grayImg[index] = (unsigned char) (0.21*ucharImg[index*3] + 0.71*ucharImg[index*3 + 1] + 0.07*ucharImg[index*3 + 2]);
	}
	
}



__global__ void hist_eq(unsigned char * deviceCharImg, float * output, float* cdf, float cdfmin, int size)
{
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	
	
	int i = tx+blockDim.x*bx;
	
	if(i < size)
	{
		deviceCharImg[i] = min(max(255*(cdf[deviceCharImg[i]] - cdfmin)/(1 - cdfmin),0.0),255.0);
		
		output[i] = (float) (deviceCharImg[i]/255.0);
		
	}
}



__global__ void histo_kernel(unsigned char * buffer, unsigned int * histo, long size)
{
	//  compute histogram with a private version in each block
	__shared__ unsigned int histo_private[HISTOGRAM_LENGTH];
	
	int bx = blockIdx.x;
	int tx = threadIdx.x;
	
	//  index of current pixel
	int index = tx+bx*blockDim.x;
	
	//  set initial values of histogram to zero 
	if (tx < HISTOGRAM_LENGTH) histo_private[tx] = 0;
	
	__syncthreads();
	
	
	int stride = blockDim.x*gridDim.x;
	
	//iterate to add values
	while (index < stride)
	{
		atomicAdd(&(histo_private[buffer[index]]), 1);
		index += stride;
	}
	
	__syncthreads();
	
	//copy private histogram to device histogram
	if(tx<256)
	{
		atomicAdd(&(histo[tx]), histo_private[tx]);
	}
}


float prob(int x, int width, int height)
{
	return 1.0*x/(width*height);
}



int main(int argc, char ** argv) {
    wbArg_t args;
    int imageWidth;
    int imageHeight;
    int imageChannels;
    wbImage_t inputImage;
    wbImage_t outputImage;
    float * hostInputImageData;
    float * hostOutputImageData;
    const char * inputImageFile;

    //@@ Insert more code here
	//  device variables
	float * deviceInputImageData;
	float * deviceOutputImageData;
	unsigned char * deviceUCharImage;
	unsigned char * deviceGrayImg;
	
	
    args = wbArg_read(argc, argv); /* parse the input arguments */

    inputImageFile = wbArg_getInputFile(args, 0);

    wbTime_start(Generic, "Importing data and creating memory on host");
    inputImage = wbImport(inputImageFile);
    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);
    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
    wbTime_stop(Generic, "Importing data and creating memory on host");

    //@@ insert code here
	hostInputImageData = wbImage_getData(inputImage);
    hostOutputImageData = wbImage_getData(outputImage);
	
	//allocate memory for device variables
	hipMalloc((void **) &deviceInputImageData, imageWidth*imageHeight*imageChannels*sizeof(float));
	hipMalloc((void **) &deviceOutputImageData, imageWidth*imageHeight*imageChannels*sizeof(float));
	hipMalloc((void **) &deviceUCharImage, imageWidth*imageHeight*imageChannels*sizeof(unsigned char));
	hipMalloc((void **) &deviceGrayImg, imageWidth*imageHeight*sizeof(unsigned char));
	
	hipMemcpy(deviceInputImageData, 
			   hostInputImageData, 
			   imageWidth*imageHeight*imageChannels*sizeof(float), 
			   hipMemcpyHostToDevice);
	
    wbLog(TRACE, "image width: ",imageWidth,", image height: ",imageHeight);
	
    //@@ insert code here
	dim3 dimBlock(12, 12, 1);
	dim3 dimGrid((imageWidth - 1)/12 + 1, (imageHeight - 1)/12 + 1, 1);
	
	//convert the image to unsigned char
	convertToChar<<<dimGrid, dimBlock>>>(deviceInputImageData, deviceUCharImage, imageWidth, imageHeight);
	
		
	//  need to convert image to grayscale
	convertToGrayScale<<<dimGrid, dimBlock>>>(deviceUCharImage, deviceGrayImg, imageWidth, imageHeight);
	
	//hipMemcpy(hostGrayImg, deviceGrayImg, imageWidth*imageHeight*sizeof(unsigned char), hipMemcpyDeviceToHost);
	
	//  allocate histogram host and set initial values of array to zero.
	unsigned int * hostHistogram;
	hostHistogram = (unsigned int *) malloc(HISTOGRAM_LENGTH*sizeof(unsigned int));
	for(int i = 0; i < HISTOGRAM_LENGTH; i++)
	{
		hostHistogram[i] = 0;
	}
	
	//  allocation for histogram from host to device
	unsigned int * deviceHistogram;
	hipMalloc((void **) &deviceHistogram,HISTOGRAM_LENGTH*sizeof(unsigned int));
	hipMemcpy(deviceHistogram, hostHistogram, HISTOGRAM_LENGTH*sizeof(unsigned int), hipMemcpyHostToDevice);

	//  size in 1D, gray image should only have one channel
	dim3 histoGrid((imageWidth*imageHeight-1)/HISTOGRAM_LENGTH + 1, 1, 1);
	dim3 histoBlock(HISTOGRAM_LENGTH,1,1);
	
	//compute the histogram
	histo_kernel<<<histoGrid, histoBlock>>>(deviceGrayImg, deviceHistogram, imageWidth*imageHeight);	
	
	//copy result back to host histogram
	hipMemcpy(hostHistogram, deviceHistogram, HISTOGRAM_LENGTH*sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	//  compute scan operation for histogram
	float * hostCDF;
	hostCDF = (float *)malloc(HISTOGRAM_LENGTH*sizeof(float));
	hostCDF[0] = prob(hostHistogram[0], imageWidth, imageHeight);
	for(int i = 1; i < HISTOGRAM_LENGTH; i++)
	{
		hostCDF[i] = hostCDF[i-1]+prob(hostHistogram[i],imageWidth,imageHeight);
	}
	
	//  compute cdfmin
	float cdfmin = hostCDF[0];
	for(int i = 1; i < HISTOGRAM_LENGTH;i++)
	{
		cdfmin = min(cdfmin, hostCDF[i]);
	}
	
	//  copy host cdf to device
	float *deviceCDF;
	hipMalloc((void **) &deviceCDF, HISTOGRAM_LENGTH*sizeof(float));
	hipMemcpy(deviceCDF, hostCDF, HISTOGRAM_LENGTH*sizeof(float), hipMemcpyHostToDevice);
	
	
	//  histogram equalization function
	
	dim3 dimGrid2((imageWidth*imageHeight*imageChannels - 1)/HISTOGRAM_LENGTH + 1, 1, 1);
	dim3 dimBlock2(HISTOGRAM_LENGTH, 1, 1);
	
	hist_eq<<<dimGrid2, dimBlock2>>>(deviceUCharImage, deviceOutputImageData, deviceCDF, cdfmin, imageWidth*imageHeight*imageChannels);
	
	//  copy results back to host
	hipMemcpy(hostOutputImageData, deviceOutputImageData, imageWidth*imageHeight*imageChannels*sizeof(float), hipMemcpyDeviceToHost);
	wbSolution(args, outputImage);
	
	hipFree(deviceUCharImage);
	hipFree(deviceGrayImg);
	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);
	
	free(hostInputImageData);
	free(hostOutputImageData);
	
	wbImage_delete(outputImage);
    wbImage_delete(inputImage);
    
	return 0;
}
