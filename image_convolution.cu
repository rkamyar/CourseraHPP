#include "hip/hip_runtime.h"
#include    <wb.h>


#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

#define Mask_width  5
#define Mask_radius Mask_width/2
#define TILE_WIDTH 12
#define BLOCK_WIDTH (TILE_WIDTH + 4)
#define w (TILE_WIDTH + Mask_width - 1)
#define clamp(x) (min(max((x), 0.0), 1.0))

//@@ INSERT CODE HERE

__global__ void convolution_2D_kernel(float *P,float *I, int height, int width, int channels, const float * __restrict__ M) 
{

	__shared__ float Ns[w][w];
	
	int k;
   for (k = 0; k < channels; k++) 
   {
	
		// loading batch
      int dest = threadIdx.y * TILE_WIDTH + threadIdx.x, destY = dest / w, destX = dest % w,srcY = blockIdx.y * TILE_WIDTH + destY - Mask_radius, srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius,
         src = (srcY * width + srcX) * channels + k;
      if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
         Ns[destY][destX] = I[src];
      else
         Ns[destY][destX] = 0;
	   
	   //load another batch
	   
	    dest = threadIdx.y * TILE_WIDTH + threadIdx.x + TILE_WIDTH * TILE_WIDTH;
		  destY = dest / w, destX = dest % w;
		  srcY = blockIdx.y * TILE_WIDTH + destY - Mask_radius;
		  srcX = blockIdx.x * TILE_WIDTH + destX - Mask_radius;
		  src = (srcY * width + srcX) * channels + k;
		  if (destY < w) 
		  {
			 if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
				Ns[destY][destX] = I[src];
			 else
				Ns[destY][destX] = 0;
		  }
	   __syncthreads();
	   
	   float output=0.0;
	    int y, x;
      for (y = 0; y < Mask_width; y++)
         for (x = 0; x < Mask_width; x++)
            output += Ns[threadIdx.y + y][threadIdx.x + x] * M[y * Mask_width + x];
     
		y = blockIdx.y * TILE_WIDTH + threadIdx.y;
      x = blockIdx.x * TILE_WIDTH + threadIdx.x;
      if (y < height && x < width)
         P[(y * width + x) * channels + k] = clamp(output);
	   
      __syncthreads();
	   
		   
   }
	
		
}
	
	
int main(int argc, char* argv[]) {
    wbArg_t args;
    int maskRows;
    int maskColumns;
    int imageChannels;
    int imageWidth;
    int imageHeight;
    char * inputImageFile;
    char * inputMaskFile;
    wbImage_t inputImage;
    wbImage_t outputImage;
    float * hostInputImageData;
    float * hostOutputImageData;
    float * hostMaskData;
    float * deviceInputImageData;
    float * deviceOutputImageData;
    float * deviceMaskData;

    args = wbArg_read(argc, argv); /* parse the input arguments */

    inputImageFile = wbArg_getInputFile(args, 0);
    inputMaskFile = wbArg_getInputFile(args, 1);

    inputImage = wbImport(inputImageFile);
    hostMaskData = (float *) wbImport(inputMaskFile, &maskRows, &maskColumns);

    assert(maskRows == 5); /* mask height is fixed to 5 in this mp */
    assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);

    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

    hostInputImageData = wbImage_getData(inputImage);
    hostOutputImageData = wbImage_getData(outputImage);

    wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

    wbTime_start(GPU, "Doing GPU memory allocation");
    hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * sizeof(float));
    wbTime_stop(GPU, "Doing GPU memory allocation");


    wbTime_start(Copy, "Copying data to the GPU");
    hipMemcpy(deviceInputImageData,
               hostInputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(deviceMaskData,
               hostMaskData,
               maskRows * maskColumns * sizeof(float),
               hipMemcpyHostToDevice);
    wbTime_stop(Copy, "Copying data to the GPU");


    wbTime_start(Compute, "Doing the computation on the GPU");
    //@@ INSERT CODE HERE
	
	dim3 dimGrid(ceil((float)imageWidth/TILE_WIDTH), ceil((float)imageHeight/TILE_WIDTH));
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	
	   
    convolution_2D_kernel<<<dimGrid, dimBlock>>>(deviceOutputImageData, deviceInputImageData, imageHeight,
                                                 imageWidth, imageChannels,
                                                 deviceMaskData);
	wbCheck( hipDeviceSynchronize() );
	
    wbTime_stop(Compute, "Doing the computation on the GPU");


    wbTime_start(Copy, "Copying data from the GPU");
    hipMemcpy(hostOutputImageData,
               deviceOutputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying data from the GPU");

    wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

    wbSolution(args, outputImage);

    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceMaskData);

    free(hostMaskData);
    wbImage_delete(outputImage);
    wbImage_delete(inputImage);

    return 0;
}
