#include "hip/hip_runtime.h"
#include	<wb.h>

#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
	int index = threadIdx.x+blockIdx.x*blockDim.x;
	if(index < len)
	{
		out[index] = in1[index]+in2[index];
	}
}

int main(int argc, char ** argv) {
	
	hipStream_t stream1, stream2, stream3, stream4;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);
	hipStreamCreate(&stream4);
	
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    
	float * dA1, * dB1, * dC1;
	float * dA2, * dB2, * dC2;
	float * dA3, * dB3, * dC3;
	float * dA4, * dB4, * dC4;
	


	
    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

	
	
	wbCheck(hipMalloc((void **) &dA1, inputLength*sizeof(float)));
	wbCheck(hipMalloc((void **) &dB1, inputLength*sizeof(float)));
	wbCheck(hipMalloc((void **) &dC1, inputLength*sizeof(float)));
	
	wbCheck(hipMalloc((void **) &dA2, inputLength*sizeof(float)));
	wbCheck(hipMalloc((void **) &dB2, inputLength*sizeof(float)));
	wbCheck(hipMalloc((void **) &dC2, inputLength*sizeof(float)));
	
	wbCheck(hipMalloc((void **) &dA3, inputLength*sizeof(float)));
	wbCheck(hipMalloc((void **) &dB3, inputLength*sizeof(float)));
	wbCheck(hipMalloc((void **) &dC3, inputLength*sizeof(float)));
	
	wbCheck(hipMalloc((void **) &dA4, inputLength*sizeof(float)));
	wbCheck(hipMalloc((void **) &dB4, inputLength*sizeof(float)));
	wbCheck(hipMalloc((void **) &dC4, inputLength*sizeof(float)));
	
	dim3 dimGrid((inputLength-1)/512 + 1, 1, 1);
	dim3 dimBlock(512, 1, 1);
	
	int SegmentSize = inputLength/4;
	
	//for improvements try using cudaDevceSynchronize();
	for(int i = 0; i < inputLength; i += SegmentSize*4)
	{
		
		//first copy input arrays to device
		//then call kernel function, passing device input and output arrays
		//copy back to host output array
		
		hipMemcpyAsync(dA1, hostInput1+i, SegmentSize*sizeof(float),hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(dB1, hostInput2+i, SegmentSize*sizeof(float),hipMemcpyHostToDevice, stream1);
		vecAdd<<<dimGrid, dimBlock, 0, stream1>>>(dA1, dB1, dC1, inputLength);
		hipMemcpyAsync(hostOutput+i, dC1, SegmentSize*sizeof(float), hipMemcpyDeviceToHost, stream1);
		
		
		hipMemcpyAsync(dA2, hostInput1+i+SegmentSize, SegmentSize*sizeof(float),hipMemcpyHostToDevice, stream2);
		hipMemcpyAsync(dB2, hostInput2+i+SegmentSize, SegmentSize*sizeof(float),hipMemcpyHostToDevice, stream2);
		vecAdd<<<dimGrid, dimBlock, 0, stream2>>>(dA2, dB2, dC2, inputLength);
		hipMemcpyAsync(hostOutput+i+SegmentSize, dC2, SegmentSize*sizeof(float), hipMemcpyDeviceToHost, stream2);
		
		
		hipMemcpyAsync(dA3, hostInput1+i+SegmentSize*2, SegmentSize*sizeof(float),hipMemcpyHostToDevice, stream3);
		hipMemcpyAsync(dB3, hostInput2+i+SegmentSize*2, SegmentSize*sizeof(float),hipMemcpyHostToDevice, stream3);
		vecAdd<<<dimGrid, dimBlock, 0, stream3>>>(dA3, dB3, dC3, inputLength);
		hipMemcpyAsync(hostOutput+i+SegmentSize*2, dC3, SegmentSize*sizeof(float), hipMemcpyDeviceToHost, stream3);
		
		
		hipMemcpyAsync(dA4, hostInput1+i+SegmentSize*3, SegmentSize*sizeof(float),hipMemcpyHostToDevice, stream4);
		hipMemcpyAsync(dB4, hostInput2+i+SegmentSize*3, SegmentSize*sizeof(float),hipMemcpyHostToDevice, stream4);
		vecAdd<<<dimGrid, dimBlock, 0, stream4>>>(dA4, dB4, dC4, inputLength);		
		hipMemcpyAsync(hostOutput+i+SegmentSize*3, dC4, SegmentSize*sizeof(float), hipMemcpyDeviceToHost, stream4);
	}
	
    wbSolution(args, hostOutput, inputLength);

	hipFree(dA1);
	hipFree(dB1);
	hipFree(dC1);
	
	hipFree(dA2);
	hipFree(dB2);
	hipFree(dC2);
	
	hipFree(dA3);
	hipFree(dB3);
	hipFree(dC3);
	
	hipFree(dA4);
	hipFree(dB4);
	hipFree(dC4);
	
    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}
